#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaNormalize.h"



// gpuNormalize
template <typename T>
__global__ void gpuNormalize( T* input, T* output, int width, int height, float scaling_factor )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	const T px = input[ y * width + x ];

	output[y*width+x] = make_float4(px.x * scaling_factor,
							  px.y * scaling_factor,
							  px.z * scaling_factor,
							  px.w * scaling_factor);
}


// cudaNormalizeRGBA
hipError_t cudaNormalizeRGBA( float4* input, const float2& input_range,
						 float4* output, const float2& output_range,
						 size_t  width,  size_t height )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0  )
		return hipErrorInvalidValue;

	const float multiplier = output_range.y / input_range.y;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	gpuNormalize<float4><<<gridDim, blockDim>>>(input, output, width, height, multiplier);

	return CUDA(hipGetLastError());
}





